#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>

#include "cutil_math.h"
#include "cxxopts.hpp"

static uint32_t width = 512;
static uint32_t height = 512;
static uint32_t samples = 128;

struct Ray {
    float3 origin;
    float3 direction;

    __host__ __device__
    Ray(float3 o_, float3 d_)
        : origin { o_ }
        , direction { d_ }
    { }

    __host__ __device__
    float3 at(float distance) const {
        return this->origin + this->direction * distance;
    }
};

enum Material_t {
    DIFF,
    SPEC,
    REFR,
};

class Sphere;

struct Intersection {
    bool valid = false;

    const Sphere* object = nullptr;

    float distance = 0;
    float3 point;
    float3 normal;
};

struct Sphere {
    float radius;
    float3 position;
    float3 emission;
    float3 colour;
    Material_t material;

    __host__ __device__
    Intersection intersect(const Ray& r) const {
        Intersection intersection {};

        intersection.object = this;

        float3 op = this->position - r.origin;
        float epsilon = 0.001f;
        float b = dot(op, r.direction);

        float disc = b * b - dot(op, op) + this->radius * this->radius;
        if (disc < 0) {
            return intersection;
        }
        else {
            disc = sqrtf(disc);
        }

        float x0 = b - disc;
        float x1 = b + disc;

        if (x0 > epsilon) {
            intersection.distance = x0;
            intersection.valid = true;
        }
        else if (x1 > epsilon) {
            intersection.distance = x1;
            intersection.valid = true;
        }

        if (intersection.valid) {
            intersection.point = r.at(intersection.distance);
            intersection.normal = normalize(intersection.point - this->position);
        }

        return intersection;
    }
};

__host__ __device__
inline Intersection intersect_scene(const Ray& r, Sphere* device_spheres, size_t sphere_count) {
    Intersection ret {};
    ret.distance = 1e20;

    for (int i = 0; i < sphere_count; i++) {
        const Sphere& sphere = device_spheres[i];

        Intersection intersection = sphere.intersect(r);
        if (intersection.valid && intersection.distance < ret.distance) {
            ret = intersection;
        }
    }

    return ret;
}

// On-device RNG from https://github.com/gz/rust-raytracer

__host__ __device__
static float getrandom(uint64_t* seed0, uint64_t* seed1) {
    *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);
    *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

    unsigned int ires = ((*seed0) << 16) + (*seed1);

    union {
        float f;
        unsigned int ui;
    } res;

    res.ui = (ires & 0x007fffff) | 0x40000000;

    return (res.f - 2.0f) / 2.0f;
}

__host__ __device__
float3 radiance(Ray& r, uint64_t* s1, uint64_t* s2, Sphere* device_spheres, size_t sphere_count) {
    float3 color_acc = make_float3(0.0f, 0.0f, 0.0f);
    float3 mask = make_float3(1.0f, 1.0f, 1.0f);

    for (int bounces = 0; bounces < 12; bounces++) {
        Intersection intersection = intersect_scene(r, device_spheres, sphere_count);
        if (!intersection.valid) {
            return make_float3(0.0f, 0.0f, 0.0f);
        }

        const Sphere& obj = *intersection.object;

        float3 x = intersection.point;
        float3 n = intersection.normal;

        float3 nl = dot(n, r.direction) < 0 ? n : n * -1;
        float3 f = obj.colour;

        color_acc += mask * obj.emission;
        float3 d;

        if (obj.material == DIFF) {
            float r1 = 2 * M_PI * getrandom(s1, s2);
            float r2 = getrandom(s1, s2);
            float r2s = sqrtf(r2);

            float3 w = nl;
            float3 u = normalize(cross(fabs(w.x) > 0.1f ? make_float3(0.0f, 1.0f, 0.0f) : make_float3(1.0f, 0.0f, 0.0f), w));
            float3 v = cross(w, u);

            d = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));
            x += nl * 0.03f;
            mask *= f;
        }
        else if (obj.material == SPEC) {
            d = r.direction - 2.0f * n * dot(n, r.direction);
            x += nl * 0.01f;
            mask *= f;
        }
        else if (obj.material == REFR) {
            bool into = dot(n, nl) > 0;
            float nc = 1.0f;
            float nt = 1.5f;
            float nnt = into ? nc / nt : nt / nc;
            float ddn = dot(r.direction, nl);
            float cos2t = 1.0f - nnt * nnt * (1.0f - ddn * ddn);

            if (cos2t < 0.0f) {
                d = reflect(r.direction, n);
                x += nl * 0.01f;
            }
            else {
                float3 tdir = normalize(r.direction * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrtf(cos2t))));

                float R0 = (nt - nc) * (nt - nc) / (nt + nc) * (nt + nc);
                float c = 1.0f - (into ? -ddn : dot(tdir, n));
                float Re = R0 + (1.0f - R0) * c * c * c * c * c;
                float Tr = 1.0f - Re;
                float P = 0.25f + 0.5f * Re;
                float RP = Re / P;
                float TP = Tr / (1.0f - P);

                if (getrandom(s1, s2) < 0.25f) {
                    mask *= RP;
                    d = reflect(r.direction, n);
                    // Reflection bias
                    x += nl * 0.02f;
                }
                else {
                    mask *= TP;
                    d = tdir;
                    // Transmission bias
                    x += nl * 0.000001f;
                }
            }
        }

        r.origin = x;
        r.direction = d;
    }

    return color_acc;
}

__host__ __device__
void render_equation(uint32_t x, uint32_t y, uint32_t width, uint32_t height, uint32_t samples, float3* output, Sphere* spheres, size_t sphere_count) {
    uint64_t i = (height - y - 1) * width + x;

    uint64_t s1 = x;
    uint64_t s2 = y;

    Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1)));

    float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f);
    float3 cy = normalize(cross(cx, cam.direction)) * .5135;
    float3 r = make_float3(0.0f);

    float alias_radius = 2.0f;

    for (int s = 0; s < samples; s++) {
        float bias_x = getrandom(&s1, &s2) - 0.5f;
        float bias_y = getrandom(&s1, &s2) - 0.5f;

        float3 d = normalize(
                cam.direction + cx * ((0.25f + x + bias_x * alias_radius) / width  - 0.5f)
                              + cy * ((0.25f + y + bias_y * alias_radius) / height - 0.5f)
        );

        Ray ray(cam.origin + normalize(d) * 40.0f, normalize(d));
        r += radiance(ray, &s1, &s2, spheres, sphere_count);
    }

    r /= samples;

    output[i] = clamp(r, 0.0f, 1.0f);
}

__global__
void render_kernel(float3* output, uint32_t width, uint32_t height, uint32_t samples, Sphere* device_spheres, size_t sphere_count) {
    __shared__ Sphere shared_spheres[128];

    if (threadIdx.x < sphere_count) {
        shared_spheres[threadIdx.x] = device_spheres[threadIdx.x];
    }
    __syncthreads();

    uint64_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t y = blockIdx.y * blockDim.y + threadIdx.y;

    render_equation(x, y, width, height, samples, output, shared_spheres, sphere_count);
}

__host__
void render_host(float3* output, Sphere* spheres, size_t sphere_count) {
    for (uint32_t x = 0; x < width; x++) {
        for (uint32_t y = 0; y < height; y++) {
            printf("\r-- Rendering pixel (%d, %d)", x, y);

            render_equation(x, y, width, height, samples, output, spheres, sphere_count);
        }
    }
    printf("\r-- Finished\n");
}

// Float to byte with gamma correction
int toInt(float n) {
    return (int)(pow(clamp(n, 0.0f, 1.0f), 1 / 2.2) * 255.0f + 0.5f);
}

void write_to_file(const char* filename, float3* buffer) {
    FILE *f = fopen(filename, "w");
    fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
    for (int i = 0; i < width * height; i++)  // loop over pixels, write RGB values
        fprintf(f, "%d %d %d ",
                toInt(buffer[i].x),
                toInt(buffer[i].y),
                toInt(buffer[i].z));
    fclose(f);
    printf("Saved image to '%s'\n", filename);
}

std::vector<Sphere> spheres {
        Sphere { 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, DIFF },
        Sphere { 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF },
        Sphere { 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF },
        Sphere { 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF },
        Sphere { 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF },
        Sphere { 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF },

        Sphere { 16.5f, { 27.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, SPEC },
        Sphere { 16.5f, { 73.0f, 16.5f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, REFR },
        Sphere { 8.0f, { 50.0f, 50.0f, 50.0f }, { 13.0f, 11.5f, 11.0f }, { 1.0f, 1.0f, 1.0f }, DIFF },
//    Sphere { 600.0f, { 50.0f, 681.6f - .77f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }
};

int main(int argc, char** argv) {
    /* Parse options */
    cxxopts::Options options(argv[0], "Path tracer for MI-PRC");

    options.add_options()
            ("w,width", "Image width", cxxopts::value<uint32_t>()->default_value("512"))
            ("h,height", "Image height", cxxopts::value<uint32_t>()->default_value("512"))
            ("s,samples", "Samples per pixel", cxxopts::value<uint32_t>()->default_value("1024"))
            ("r,random", "Random spheres", cxxopts::value<uint32_t>()->default_value("0"))
            ("o,output", "Output file", cxxopts::value<std::string>())
            ;

    auto opts = options.parse(argc, argv);

    width = opts["w"].as<uint32_t>();
    height = opts["h"].as<uint32_t>();
    samples = opts["s"].as<uint32_t>();

    /* Add random spheres */
    std::random_device rd;
    std::default_random_engine gen(rd());
    std::uniform_real_distribution<float> dist(-20.0f, 20.0f);

    for (int i = 0; i < opts["r"].as<uint32_t>(); i++) {
        float3 center = make_float3(45.0f, 24.0f, 65.0f);
        Sphere s { 4.0f, center + make_float3(dist(gen), dist(gen), dist(gen)), { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF };
        spheres.push_back(s);
    }

    /* Allocate memory */
    float3* output_h = new float3[width * height];
    float3* output_d;

    hipMalloc(&output_d, width * height * sizeof(float3));

    Sphere* device_spheres;
    hipMalloc(&device_spheres, spheres.size() * sizeof(Sphere));
    hipMemcpy(device_spheres, spheres.data(), spheres.size() * sizeof(Sphere), hipMemcpyHostToDevice);

    dim3 block(32, 32, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    /* Render */
    printf("Rendering...\n");

#ifdef USE_CPU
    render_host(output_h, spheres.data(), spheres.size());
#else
    render_kernel<<<grid, block>>>(output_d, width, height, samples, device_spheres, spheres.size());
#endif

    hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);

    /* Write to file */
    if (opts.count("o") > 0) {
        write_to_file(opts["o"].as<std::string>().c_str(), output_h);
    }

    /* Free memory */
    hipFree(output_d);
    hipFree(device_spheres);

    delete[] output_h;
}
